#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
using std::cout;
using std::endl;
using std::vector;

#include "helper.h"
#include "parameter.h"
#include "globals.h"

#include "poly_arithmetic.cuh"
#include "fips202.h"
#include "pack_unpack.h"


#include <stdlib.h>
#include <time.h>
//srand ((unsigned int) time (NULL));

void random_bytes(uint8_t  seed[], size_t num_bytes)
{
    size_t i;

    for (i = 0; i < num_bytes; i++)
    {
        seed[i] = rand ();
    }

}


void gen_matrix(uint16_t A[L][L][N], uint8_t seed[SABER_SEEDBYTES]) {
    uint8_t buf[L * L * 32 * EQ];
    int i;
    shake128(buf, sizeof(buf), seed, SABER_SEEDBYTES);
    for (i = 0; i < L; i++)
    {
        BS2POLVECq(buf + i * L * 32 * EQ, A[i]);
    }

}

void gen_secret(uint16_t s[L][N], const uint8_t seed[SABER_NOISE_SEEDBYTES])
{
    uint8_t buf[L * SABER_POLYCOINBYTES];
    size_t i;

    shake128(buf, sizeof(buf), seed, SABER_NOISE_SEEDBYTES);

    for (i = 0; i < L; i++)
    {
        cbd(s[i], buf + i * SABER_POLYCOINBYTES);
    }
}


 __global__ void poly_mul(uint16_t c[], uint16_t a[], uint16_t b[], uint16_t q)
{
    register int i = blockIdx.x * 1 + threadIdx.x;

    uint16_t rc = a[i] * b[i];

    c[i] = rc & (q-1);
}

__global__ void vector_mul(uint16_t **c, uint16_t **a, uint16_t **b, uint16_t q)
{
    register int i = blockIdx.x * 1 + threadIdx.x;


    poly_mul<<<1,N,0,0>>>(c[i],a[i],b[i],q);

}

void matrix_vector_mul(uint16_t **b, uint16_t ***A, uint16_t **s) {

    for (int i = 0; i < L; i++){
        vector_mul<<<1,L,0,0>>>(b, A[i], s, 1<<13);

    }

}

void KeyGen(uint8_t pk[SABER_INDCPA_PUBLICKEYBYTES], uint8_t sk[SABER_INDCPA_SECRETKEYBYTES]) {
    uint16_t A[L][L][N];
    uint16_t s[L][N];
    uint16_t b[L][N] = {0};

    uint8_t seed_A[SABER_SEEDBYTES];
    uint8_t seed_s[SABER_NOISE_SEEDBYTES];

    random_bytes(seed_A,SABER_SEEDBYTES);
    random_bytes(seed_s,SABER_NOISE_SEEDBYTES);

    gen_matrix(A, seed_A);
    gen_secret(s, seed_s);

    uint16_t ***A_device = (uint16_t***)malloc(sizeof (uint16_t**) * L);
    for (int i = 0; i<L;i++)
        A_device[i] = (uint16_t**)malloc(sizeof (uint16_t*) * L);

    for (int i = 0; i<L;i++){
        for (int j = 0; j<L;j++)
             hipMalloc(&A_device[i][j], sizeof (uint16_t) * N);
    }
    hipMemcpy(A_device,A,sizeof (uint16_t) * N*L*L,hipMemcpyHostToDevice);

    uint16_t **s_device = (uint16_t**)malloc(sizeof (uint16_t*) * L);
    for (int i = 0; i<L;i++)
        hipMalloc(&s_device[i], sizeof (uint16_t) * N);

    uint16_t **b_device = (uint16_t**)malloc(sizeof (uint16_t*) * L);
    for (int i = 0; i<L;i++)
        hipMalloc(&b_device[i], sizeof (uint16_t) * N);

    hipMemcpy(A_device,A,sizeof (uint16_t) * N*L*L,hipMemcpyHostToDevice);

    matrix_vector_mul(b_device, A_device, s_device);

    free(A);
    hipFree(A_device);




}

int main() {
    uint8_t pk[SABER_INDCPA_PUBLICKEYBYTES];
    uint8_t sk[SABER_INDCPA_SECRETKEYBYTES];

    KeyGen(pk,sk);

    return 0;
}
